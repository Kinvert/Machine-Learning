#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <cassert>
#include <algorithm>

#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipDNN.h"

#include "mnist.h"

#define checkCUDNN(expression)                             \
{                                                         \
    hipdnnStatus_t status = (expression);                  \
    if (status != HIPDNN_STATUS_SUCCESS) {                 \
        printf("Error on line %d: %s\n", __LINE__, #expression); \
        exit(1);                                           \
    }                                                     \
}

#define checkCudaErrors(expression)                        \
{                                                         \
    hipError_t err = (expression);                       \
    if (err != hipSuccess) {                             \
        printf("Error on line %d: %s\n", __LINE__, #expression); \
        exit(1);                                           \
    }                                                     \
}

#define checkCublasErrors(expression)                      \
{                                                         \
    hipblasStatus_t status = (expression);                 \
    if (status != HIPBLAS_STATUS_SUCCESS) {                \
        printf("Error on line %d: %s\n", __LINE__, #expression); \
        exit(1);                                           \
    }                                                     \
}

#define NUM_CLASSES 10
#define IMAGE_SIZE 784

#define NUM_TRAINING_IMAGES 60000
#define NUM_TEST_IMAGES 10000

#define BATCH_SIZE 128

#define NUM_EPOCHS 10

#define LEARNING_RATE 0.01

// Helper function to set the dimensions of a 4D tensor
void setTensor4dDesc(hipdnnTensorDescriptor_t& tensorDesc, int n, int c, int h, int w)
{
    checkCUDNN(hipdnnSetTensor4dDescriptor(tensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w));
}

// Helper function to set the dimensions of a 2D tensor
void setTensor2dDesc(hipdnnTensorDescriptor_t& tensorDesc, int n, int c)
{
    checkCUDNN(hipdnnSetTensor4dDescriptor(tensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, 1, 1));
}

// Helper function to create a filter descriptor
void createFilterDesc(hipdnnFilterDescriptor_t& filterDesc, int n, int c, int h, int w)
{
    checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, n, c, h, w));
}

int main(int argc, char** argv)
{
    // Load MNIST dataset
    mnist_data* trainingData = NULL;
    unsigned int numTrainingImages = 0;
    mnist_data* testData = NULL;
    unsigned int numTestImages = 0;
    if (mnistLoad(&trainingData, &numTrainingImages, &testData, &numTestImages, "../../../zData/MNIST/train-images-idx3-ubyte.gz", "../../../zData/MNIST/train-labels-idx1-ubyte.gz", "../../../zData/MNIST/t10k-images-idx3-ubyte.gz", "../../../zData/MNIST/t10k-labels-idx1-ubyte.gz"))
    {
        printf("Error loading MNIST dataset.\n");
        return 1;
    }

    // Create CUDA objects
    hipStream_t stream;
    checkCudaErrors(hipStreamCreate(&stream));
    hipblasHandle_t cublasHandle;
    checkCublasErrors(hipblasCreate(&cublasHandle));
    hipdnnHandle_t cudnnHandle;
    checkCUDNN(hipdnnCreate(&cudnnHandle));

    // Allocate device memory for inputs and outputs
    float* dInputImages;
    checkCudaErrors(hipMalloc((void**)&dInputImages, BATCH_SIZE * IMAGE_SIZE * sizeof(float)));
    float* dOutputLabels;
    checkCudaErrors(hipMalloc((void**)&dOutputLabels, BATCH_SIZE * NUM_CLASSES * sizeof(float)));
    float* dOutputLogits;
    checkCudaErrors(hipMalloc((void**)&dOutputLogits, BATCH_SIZE * NUM_CLASSES * sizeof(float)));

    // Create tensor descriptors for input and output
    hipdnnTensorDescriptor_t inputTensorDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensorDesc));
    setTensor2dDesc(inputTensorDesc, BATCH_SIZE, IMAGE_SIZE);
    hipdnnTensorDescriptor_t outputTensorDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensorDesc));
    setTensor2dDesc(outputTensorDesc, BATCH_SIZE, NUM_CLASSES);

    // Create filter descriptor for fully connected layer weights
    hipdnnFilterDescriptor_t filterDesc;
    createFilterDesc(filterDesc, NUM_CLASSES, IMAGE_SIZE, 1, 1);

    // Allocate device memory for fully connected layer weights and biases
    float* dFcWeights;
    checkCudaErrors(hipMalloc((void**)&dFcWeights, NUM_CLASSES * IMAGE_SIZE * sizeof(float)));
    float* dFcBiases;
    checkCudaErrors(hipMalloc((void**)&dFcBiases, NUM_CLASSES * sizeof(float)));

    // Allocate host memory for fully connected layer weights and biases
    float* hFcWeights = (float*)malloc(NUM_CLASSES * IMAGE_SIZE * sizeof(float));
    float* hFcBiases = (float*)malloc(NUM_CLASSES * sizeof(float));

    // Initialize fully connected layer weights and biases to random values
    for (int i = 0; i < NUM_CLASSES * IMAGE_SIZE; i++)
    {
        hFcWeights[i] = (float)rand() / RAND_MAX - 0.5f;
    }
    for (int i = 0; i < NUM_CLASSES; i++)
    {
        hFcBiases[i] = (float)rand() / RAND_MAX - 0.5f;
    }

    // Copy fully connected layer weights and biases to device memory
    checkCudaErrors(hipMemcpy(dFcWeights, hFcWeights, NUM_CLASSES * IMAGE_SIZE * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dFcBiases, hFcBiases, NUM_CLASSES * sizeof(float), hipMemcpyHostToDevice));

    // Create convolution descriptor for fully connected layer
    hipdnnConvolutionDescriptor_t convDesc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // Compute fully connected layer output tensor dimensions
    int fcOutputDim[4];
    checkCUDNN(cudnnGetConvolutionNdForwardOutputDim(convDesc, inputTensorDesc, filterDesc, 4, fcOutputDim));

    // Create tensor descriptor for fully connected layer output
    hipdnnTensorDescriptor_t fcOutputTensorDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&fcOutputTensorDesc));
    setTensor4dDesc(fcOutputTensorDesc, fcOutputDim[0], fcOutputDim[1], fcOutputDim[2], fcOutputDim[3]);

    // Allocate device memory for fully connected layer output
    float* dFcOutput;
    checkCudaErrors(hipMalloc((void**)&dFcOutput, BATCH_SIZE * NUM_CLASSES * sizeof(float)));

    // Allocate device memory for fully connected layer gradients
    float* dFcOutputGrad;
    checkCudaErrors(hipMalloc((void**)&dFcOutputGrad, BATCH_SIZE * NUM_CLASSES * sizeof(float)));
    float* dFcWeightsGrad;
    checkCudaErrors(hipMalloc((void**)&dFcWeightsGrad, NUM_CLASSES * IMAGE_SIZE * sizeof(float)));
    float* dFcBiasesGrad;
    checkCudaErrors(hipMalloc((void**)&dFcBiasesGrad, NUM_CLASSES * sizeof(float)));

    // Allocate host memory for gradients
    float* hFcWeightsGrad = (float*)malloc(NUM_CLASSES * IMAGE_SIZE * sizeof(float));
    float* hFcBiasesGrad = (float*)malloc(NUM_CLASSES * sizeof(float));

    // Create operation descriptor for fully connected layer forward propagation
    hipdnnConvolutionFwdAlgo_t fcFwdAlgo;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle, inputTensorDesc, filterDesc, convDesc, fcOutputTensorDesc, HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, 0, &fcFwdAlgo));

    // Allocate device memory for fully connected layer forward propagation workspace
    void* dFcFwdWorkspace;
    size_t fcFwdWorkspaceSize;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle, inputTensorDesc, filterDesc, convDesc, fcOutputTensorDesc, fcFwdAlgo, &fcFwdWorkspaceSize));
    checkCudaErrors(hipMalloc((void**)&dFcFwdWorkspace, fcFwdWorkspaceSize));

    // Create operation descriptor for fully connected layer backward data propagation
    hipdnnConvolutionBwdDataAlgo_t fcBwdDataAlgo;
    checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cudnnHandle, filterDesc, fcOutputTensorDesc, convDesc, inputTensorDesc, HIPDNN_CONVOLUTION_BWD_DATA_SPECIFY_WORKSPACE_LIMIT, 0, &fcBwdDataAlgo));

    // Allocate device memory for fully connected layer backward data propagation workspace
    void* dFcBwdDataWorkspace;
    size_t fcBwdDataWorkspaceSize;
    checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnnHandle, filterDesc, fcOutputTensorDesc, convDesc, inputTensorDesc, fcBwdDataAlgo, &fcBwdDataWorkspaceSize));
    checkCudaErrors(hipMalloc((void**)&dFcBwdDataWorkspace, fcBwdDataWorkspaceSize));

    // Create operation descriptor for fully connected layer backward filter propagation
    hipdnnConvolutionBwdFilterAlgo_t fcBwdFilterAlgo;
    checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnnHandle, inputTensorDesc, fcOutputTensorDesc, convDesc, filterDesc, HIPDNN_CONVOLUTION_BWD_FILTER_SPECIFY_WORKSPACE_LIMIT, 0, &fcBwdFilterAlgo));

    // Allocate device memory for fully connected layer backward filter propagation workspace
    void* dFcBwdFilterWorkspace;
    size_t fcBwdFilterWorkspaceSize;
    checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnnHandle, inputTensorDesc, fcOutputTensorDesc, convDesc, filterDesc, fcBwdFilterAlgo, &fcBwdFilterWorkspaceSize));
    checkCudaErrors(hipMalloc((void**)&dFcBwdFilterWorkspace, fcBwdFilterWorkspaceSize));

    // Create softmax algorithm descriptor
    hipdnnSoftmaxAlgorithm_t softmaxAlgorithm;
    checkCUDNN(cudnnGetSoftmaxAlgorithm(cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &softmaxAlgorithm));

    // Create softmax mode descriptor
    hipdnnSoftmaxMode_t softmaxMode;
    checkCUDNN(cudnnGetSoftmaxMode(cudnnHandle, &softmaxMode));

    // Create cross-entropy loss descriptor
    cudnnCrossEntropyLossDescriptor_t lossDesc;
    checkCUDNN(cudnnCreateCrossEntropyLossDescriptor(&lossDesc));

    // Train network
    for (int epoch = 0; epoch < NUM_EPOCHS; epoch++)
    {
        for (int batch = 0; batch < NUM_TRAINING_IMAGES / BATCH_SIZE; batch++)
        {
            // Copy input images and output labels to device memory
            checkCudaErrors(hipMemcpyAsync(dInputImages, trainingData[batch * BATCH_SIZE].image, BATCH_SIZE * IMAGE_SIZE * sizeof(float), hipMemcpyHostToDevice, stream));
            checkCudaErrors(hipMemcpyAsync(dOutputLabels, trainingData[batch * BATCH_SIZE].label, BATCH_SIZE * NUM_CLASSES * sizeof(float), hipMemcpyHostToDevice, stream));

            // Forward propagate through fully connected layer
            checkCUDNN(hipdnnConvolutionForward(cudnnHandle, &LEARNING_RATE, inputTensorDesc, dInputImages, filterDesc, dFcWeights, convDesc, fcFwdAlgo, dFcFwdWorkspace, fcFwdWorkspaceSize, &LEARNING_RATE, fcOutputTensorDesc, dFcOutput));

            // Add biases to fully connected layer output
            checkCublasErrors(hipblasSgeam(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, NUM_CLASSES, BATCH_SIZE, &LEARNING_RATE, dFcBiases, NUM_CLASSES, &LEARNING_RATE, dFcOutput, NUM_CLASSES, dFcOutput, NUM_CLASSES));

            // Perform softmax on fully connected layer output
            checkCUDNN(hipdnnSoftmaxForward(cudnnHandle, softmaxAlgorithm, softmaxMode, &LEARNING_RATE, fcOutputTensorDesc, dFcOutput, &LEARNING_RATE, fcOutputTensorDesc, dOutputLogits));

            // Compute cross-entropy loss
            float loss;
            checkCUDNN(cudnnCrossEntropyLoss(cudnnHandle, lossDesc, fcOutputTensorDesc, dOutputLogits, outputTensorDesc, dOutputLabels, &loss));

            // Backward propagate through fully connected layer
            checkCUDNN(hipdnnSoftmaxBackward(cudnnHandle, softmaxAlgorithm, softmaxMode, &LEARNING_RATE, fcOutputTensorDesc, dOutputLogits, outputTensorDesc, dOutputLabels, &LEARNING_RATE, fcOutputTensorDesc, dFcOutputGrad));
            checkCUDNN(hipdnnConvolutionBackwardBias(cudnnHandle, &LEARNING_RATE, fcOutputTensorDesc, dFcOutputGrad, &LEARNING_RATE, outputTensorDesc, dFcBiasesGrad));
            checkCUDNN(hipdnnConvolutionBackwardFilter(cudnnHandle, &LEARNING_RATE, inputTensorDesc, dInputImages, fcOutputTensorDesc, dFcOutputGrad, convDesc, fcBwdFilterAlgo, dFcBwdFilterWorkspace, fcBwdFilterWorkspaceSize, &LEARNING_RATE, filterDesc, dFcWeightsGrad));
            checkCUDNN(hipdnnConvolutionBackwardData(cudnnHandle, &LEARNING_RATE, filterDesc, dFcWeights, fcOutputTensorDesc, dFcOutputGrad, convDesc, fcBwdDataAlgo, dFcBwdDataWorkspace, fcBwdDataWorkspaceSize, &LEARNING_RATE, inputTensorDesc, dInputImages));

            // Update fully connected layer weights and biases
            checkCublasErrors(hipblasSaxpy(cublasHandle, NUM_CLASSES * IMAGE_SIZE, &LEARNING_RATE, dFcWeightsGrad, 1, dFcWeights, 1));
            checkCublasErrors(hipblasSaxpy(cublasHandle, NUM_CLASSES, &LEARNING_RATE, dFcBiasesGrad, 1, dFcBiases, 1));
        }
    }

    // Copy fully connected layer weights and biases back to host memory
    checkCudaErrors(hipMemcpy(hFcWeights, dFcWeights, NUM_CLASSES * IMAGE_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(hFcBiases, dFcBiases, NUM_CLASSES * sizeof(float), hipMemcpyDeviceToHost));

    // Test network
    for (int i = 0; i < numTestImages; i++)
    {
        // Copy input image to device memory
        checkCudaErrors(hipMemcpyAsync(dInputImages, testData[i].image, IMAGE_SIZE * sizeof(float), hipMemcpyHostToDevice, stream));

        // Forward propagate through fully connected layer
        checkCUDNN(hipdnnConvolutionForward(cudnnHandle, &LEARNING_RATE, inputTensorDesc, dInputImages, filterDesc, dFcWeights, convDesc, fcFwdAlgo, dFcFwdWorkspace, fcFwdWorkspaceSize, &LEARNING_RATE, fcOutputTensorDesc, dFcOutput));

        // Add biases to fully connected layer output
        checkCublasErrors(hipblasSgeam(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, NUM_CLASSES, BATCH_SIZE, &LEARNING_RATE, dFcBiases, NUM_CLASSES, &LEARNING_RATE, dFcOutput, NUM_CLASSES, dFcOutput, NUM_CLASSES));

        // Perform softmax on fully connected layer output
        checkCUDNN(hipdnnSoftmaxForward(cudnnHandle, softmaxAlgorithm, softmaxMode, &LEARNING_RATE, fcOutputTensorDesc, dFcOutput, &LEARNING_RATE, fcOutputTensorDesc, dOutputLogits));

        // Find index of maximum value in output logits
        int maxIndex = 0;
        float maxValue = dOutputLogits[0];
        for (int j = 1; j < NUM_CLASSES; j++)
        {
            if (dOutputLogits[j] > maxValue)
            {
                maxIndex = j;
                maxValue = dOutputLogits[j];
            }
        }

        // Check if maximum value is correct
        if (maxIndex != testData[i].label)
        {
            printf("Test image %d: Incorrect classification\n", i);
        }
    }

    // Destroy resources
    checkCudaErrors(hipFree(dInputImages));
    checkCudaErrors(hipFree(dOutputLabels));
    checkCudaErrors(hipFree(dFcOutput));
    checkCudaErrors(hipFree(dFcOutputGrad));
    checkCudaErrors(hipFree(dFcWeights));
    checkCudaErrors(hipFree(dFcWeightsGrad));
    checkCudaErrors(hipFree(dFcBiases));
    checkCudaErrors(hipFree(dFcBiasesGrad));
    checkCudaErrors(hipFree(dFcFwdWorkspace));
    checkCudaErrors(hipFree(dFcBwdDataWorkspace));
    checkCudaErrors(hipFree(dFcBwdFilterWorkspace));
    checkCudaErrors(hipFree(dOutputLogits));
    checkCudaErrors(hipStreamDestroy(stream));
    checkCublasErrors(hipblasDestroy(cublasHandle));
    checkCUDNN(hipdnnDestroy(c






