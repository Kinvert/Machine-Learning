#include <hipblas.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define IMAGE_WIDTH 28
#define IMAGE_HEIGHT 28
#define IMAGE_SIZE IMAGE_WIDTH * IMAGE_HEIGHT
#define NUM_CLASSES 10
#define NUM_CHANNELS 1
#define BATCH_SIZE 64
#define NUM_EPOCHS 10
#define LEARNING_RATE 0.001f

struct TrainingData
{
    float image[IMAGE_SIZE];
    float label[NUM_CLASSES];
};

struct TestData
{
    float image[IMAGE_SIZE];
    float label[NUM_CLASSES];
};

int32_t readInt(std::ifstream& stream)
{
    int32_t value;
    stream.read((char*)&value, 4);
    return __builtin_bswap32(value);
}

hipdnnSoftmaxMode_t softmaxMode = HIPDNN_SOFTMAX_ACCURATE;
hipdnnSoftmaxAlgorithm_t softmaxAlgorithm = CUDNN_SOFTMAX_ALGORITHM_ACCURATE;


// Load MNIST training data
int loadTrainingData(TrainingData* trainingData)
{
    // Open the training images file
    std::ifstream file("../../../zData/MNIST/train-images-idx3-ubyte", std::ios::binary);

    // Read the magic number and number of images
    int32_t magic = readInt(file);
    int32_t numImages = readInt(file);

    // Read the image data
    float* hTrainingData = new float[NUM_IMAGES * IMAGE_SIZE];
    for (int i = 0; i < NUM_IMAGES; i++)
    {
        for (int j = 0; j < IMAGE_SIZE; j++)
        {
            uint8_t pixel;
            file.read((char*)&pixel, 1);
            hTrainingData[i * IMAGE_SIZE + j] = pixel / 255.0f;
        }
    }

    // Close the file
    file.close();

    // Open training labels file
    std::ifstream file("../../../zData/MNIST/train-labels-idx1-ubyte", std::ios::binary);
    if (!file.is_open())
    {
        std::printf("Error opening file\n");
        return 1;
    }

    // Read header
    file.read((char*)&magicNumber, 4);
    file.read((char*)&numImages, 4);
    magicNumber = __builtin_bswap32(magicNumber);
    numImages = __builtin_bswap32(numImages);

    // Read data
    for (int i = 0; i < numImages; i++)
    {
        unsigned char label;
        file.read((char*)&label, 1);
        trainingData[i].label[(int)label] = 1.0f;
    }

    file.close();

    return 0;
}

// Load MNIST test data
int loadTestData(TestData* testData)
{
    // Open test images file
    std::FILE* file = std::fopen("../../../zData/MNIST/t10k-images-idx3-ubyte", "rb");
    if (file == NULL)
    {
        std::printf("Error opening file\n");
        return 1;
    }

    // Read header
    int magicNumber;
    int numImages;
    int imageRows;
    int imageCols;
    std::fread((char*)&magicNumber, 4, 1, file);
    std::fread((char*)&numImages, 4, 1, file);
    std::fread((char*)&imageRows, 4, 1, file);
    std::fread((char*)&imageCols, 4, 1, file);
    magicNumber = __builtin_bswap32(magicNumber);
    numImages = __builtin_bswap32(numImages);
    imageRows = __builtin_bswap32(imageRows);
    imageCols = __builtin_bswap32(imageCols);

    // Read data
    for (int i = 0; i < numImages; i++)
    {
        for (int j = 0; j < IMAGE_SIZE; j++)
        {
            unsigned char pixel;
            std::fread((char*)&pixel, 1, 1, file);
            testData[i].image[j] = (float)pixel / 255.0f;
        }
    }

    std::fclose(file);

    // Open test labels file
    file = std::fopen("../../../zData/MNIST/t10k-labels-idx1-ubyte", "rb");
    if (file == NULL)
    {
        std::printf("Error opening file\n");
        return 1;
    }

    // Read header
    std::fread((char*)&magicNumber, 4, 1, file);
    std::fread((char*)&numImages, 4, 1, file);
    magicNumber = __builtin_bswap32(magicNumber);
    numImages = __builtin_bswap32(numImages);

    // Read data
    for (int i = 0; i < numImages; i++)
    {
        unsigned char label;
        std::fread((char*)&label, 1, 1, file);
        testData[i].label[(int)label] = 1.0f;
    }

    std::fclose(file);

    return 0;
}

// Check for CUDA errors
void checkCudaErrors(hipError_t error)
{
    if (error != hipSuccess)
    {
        std::printf("CUDA error: %s\n", hipGetErrorString(error));
        std::exit(1);
    }
}

// Check for CUDNN errors
void checkCUDNN(hipdnnStatus_t status)
{
    if (status != HIPDNN_STATUS_SUCCESS)
    {
        std::printf("CUDNN error: %s\n", hipdnnGetErrorString(status));
        std::exit(1);
    }
}

int main(int argc, char** argv)
{
    // Seed random number generator
    std::srand((unsigned int)time(NULL));

    // Initialize CUDA
    checkCudaErrors(hipSetDevice(0));

    // Allocate host memory
    TrainingData* hTrainingData = (TrainingData*)std::malloc(sizeof(TrainingData) * 60000);
    TestData* hTestData = (TestData*)std::malloc(sizeof(TestData) * 10000);
    float* hFcWeights = (float*)std::malloc(sizeof(float) * IMAGE_SIZE * NUM_CLASSES);
    float* hFcBiases = (float*)std::malloc(sizeof(float) * NUM_CLASSES);
    float* hFcOutput = (float*)std::malloc(sizeof(float) * BATCH_SIZE * NUM_CLASSES);
    float* hFcOutputGrad = (float*)std::malloc(sizeof(float) * BATCH_SIZE * NUM_CLASSES);
    float* hSoftmaxOutput = (float*)std::malloc(sizeof(float) * BATCH_SIZE * NUM_CLASSES);
    float* hSoftmaxOutputGrad = (float*)std::malloc(sizeof(float) * BATCH_SIZE * NUM_CLASSES);
    float* hSoftmaxLoss = (float*)std::malloc(sizeof(float) * BATCH_SIZE);

    // Allocate device memory
    TrainingData* dTrainingData;
    TestData* dTestData;
    float* dFcWeights;
    float* dFcBiases;
    float* dFcOutput;
    float* dFcOutputGrad;
    float* dSoftmaxOutput;
    float* dSoftmaxOutputGrad;
    float* dSoftmaxLoss;
    checkCudaErrors(hipMalloc((void**)&dTrainingData, sizeof(TrainingData) * 60000));
    checkCudaErrors(hipMalloc((void**)&dTestData, sizeof(TestData) * 10000));
    checkCudaErrors(hipMalloc((void**)&dFcWeights, sizeof(float) * IMAGE_SIZE * NUM_CLASSES));
    checkCudaErrors(hipMalloc((void**)&dFcBiases, sizeof(float) * NUM_CLASSES));
    checkCudaErrors(hipMalloc((void**)&dFcOutput, sizeof(float) * BATCH_SIZE * NUM_CLASSES));
    checkCudaErrors(hipMalloc((void**)&dFcOutputGrad, sizeof(float) * BATCH_SIZE * NUM_CLASSES));
    checkCudaErrors(hipMalloc((void**)&dSoftmaxOutput, sizeof(float) * BATCH_SIZE * NUM_CLASSES));
    checkCudaErrors(hipMalloc((void**)&dSoftmaxOutputGrad, sizeof(float) * BATCH_SIZE * NUM_CLASSES));
    checkCudaErrors(hipMalloc((void**)&dSoftmaxLoss, sizeof(float) * BATCH_SIZE));

    // Initialize CUDNN
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));
    
    // Create CUDNN tensors
    hipdnnTensorDescriptor_t inputTensor;
    hipdnnTensorDescriptor_t outputTensor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
    checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, BATCH_SIZE, NUM_CHANNELS, IMAGE_HEIGHT, IMAGE_WIDTH));
    checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, BATCH_SIZE, NUM_CLASSES, 1, 1));

    // Create CUDNN filter descriptor
    hipdnnFilterDescriptor_t filterDesc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, NUM_CLASSES, NUM_CHANNELS, IMAGE_HEIGHT, IMAGE_WIDTH));

    // Create CUDNN convolution descriptor
    hipdnnConvolutionDescriptor_t convDesc;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, 0, 0, 1, 1, 1, 1, HIPDNN_CONVOLUTION));

    // Create CUDNN convolution operation
    hipdnnConvolutionFwdAlgo_t convFwdAlgo;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, inputTensor, filterDesc, convDesc, outputTensor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &convFwdAlgo));
    hipdnnConvolutionBwdDataAlgo_t convBwdDataAlgo;
    checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cudnn, filterDesc, outputTensor, convDesc, inputTensor, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &convBwdDataAlgo));
    hipdnnConvolutionBwdFilterAlgo_t convBwdFilterAlgo;
    checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn, inputTensor, outputTensor, convDesc, filterDesc, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &convBwdFilterAlgo));
    size_t convFwdWorkspaceSize;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, inputTensor, filterDesc, convDesc, outputTensor, convFwdAlgo, &convFwdWorkspaceSize));
    size_t convBwdDataWorkspaceSize;
    checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, filterDesc, outputTensor, convDesc, inputTensor, convBwdDataAlgo, &convBwdDataWorkspaceSize));
    size_t convBwdFilterWorkspaceSize;
    checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn, inputTensor, outputTensor, convDesc, filterDesc, convBwdFilterAlgo, &convBwdFilterWorkspaceSize));

    // Allocate device memory for convolution workspace
    void* dConvFwdWorkspace;
    void* dConvBwdDataWorkspace;
    void* dConvBwdFilterWorkspace;
    checkCudaErrors(hipMalloc((void**)&dConvFwdWorkspace, convFwdWorkspaceSize));
    checkCudaErrors(hipMalloc((void**)&dConvBwdDataWorkspace, convBwdDataWorkspaceSize));
    checkCudaErrors(hipMalloc((void**)&dConvBwdFilterWorkspace, convBwdFilterWorkspaceSize));

    // Train network
    for (int epoch = 0; epoch < NUM_EPOCHS; epoch++)
    {
        for (int batch = 0; batch < NUM_BATCHES; batch++)
        {
            // Select training data for batch
            int offset = batch * BATCH_SIZE;
            TrainingData* batchTrainingData = dTrainingData + offset;

            // Forward pass
            checkCudaErrors(hipMemcpy(dFcInput, batchTrainingData, sizeof(TrainingData) * BATCH_SIZE, hipMemcpyDeviceToDevice));
            checkCUDNN(hipdnnConvolutionForward(cudnn, (void*)&alpha, inputTensor, dFcInput, filterDesc, dFcWeights, convDesc, convFwdAlgo, dConvFwdWorkspace, (void*)&beta, outputTensor, dFcOutput));
            checkCUDNN(hipdnnAddTensor(cudnn, (void*)&alpha, outputTensor, dFcBiases, (void*)&alpha, outputTensor, dFcOutput));
            checkCUDNN(hipdnnSoftmaxForward(cudnn, softmaxAlgorithm, softmaxMode, (void*)&alpha, outputTensor, dFcOutput, (void*)&beta, outputTensor, dSoftmaxOutput));

            // Compute loss
            checkCUDNN(cudnnSoftmaxCrossEntropyLoss(cudnn, (void*)&alpha, outputTensor, batchTrainingData->label, (void*)&beta, outputTensor, dSoftmaxLoss));

            // Backward pass
            checkCUDNN(hipdnnSoftmaxBackward(cudnn, softmaxAlgorithm, softmaxMode, (void*)&alpha, outputTensor, dSoftmaxOutput, outputTensor, dSoftmaxOutputGrad, (void*)&beta, outputTensor, dFcOutputGrad));
            checkCUDNN(hipdnnConvolutionBackwardFilter(cudnn, (void*)&alpha, inputTensor, dFcInput, outputTensor, dFcOutputGrad, convDesc, convBwdFilterAlgo, dConvBwdFilterWorkspace, (void*)&beta, filterDesc, dFcWeights));
            checkCUDNN(hipdnnConvolutionBackwardData(cudnn, (void*)&alpha, filterDesc, dFcWeights, outputTensor, dFcOutputGrad, convDesc, convBwdDataAlgo, dConvBwdDataWorkspace, (void*)&beta, inputTensor, dFcInput));

            // Update weights and biases
            float learningRate = 0.001f;
            checkCudaErrors(hipblasSaxpy(cublas, NUM_CLASSES, (void*)&learningRate, dFcOutputGrad, 1, dFcBiases, 1));
            checkCudaErrors(hipblasSgemv(cublas, HIPBLAS_OP_T, IMAGE_SIZE, NUM_CLASSES, (void*)&learningRate, dFcInput, IMAGE_SIZE, dFcOutputGrad, 1, (void*)&alpha, dFcWeights, 1));
        }
    }

    // Test network
    int numCorrect = 0;
    for (int i = 0; i < NUM_TEST_IMAGES; i++)
    {
        // Forward pass
        checkCudaErrors(hipMemcpy(dFcInput, dTestData + i, sizeof(TestData), hipMemcpyDeviceToDevice));
        checkCUDNN(hipdnnConvolutionForward(cudnn, (void*)&alpha, inputTensor, dFcInput, filterDesc, dFcWeights, convDesc, convFwdAlgo, dConvFwdWorkspace, (void*)&beta, outputTensor, dFcOutput));
        checkCUDNN(hipdnnAddTensor(cudnn, (void*)&alpha, outputTensor, dFcBiases, (void*)&alpha, outputTensor, dFcOutput));
        checkCUDNN(hipdnnSoftmaxForward(cudnn, softmaxAlgorithm, softmaxMode, (void*)&alpha, outputTensor, dFcOutput, (void*)&beta, outputTensor, dSoftmaxOutput));

        // Select class with maximum probability
        int classIdx;
        float classProb;
        checkCudaErrors(hipMemcpy(&classProb, dSoftmaxOutput, sizeof(float), hipMemcpyDeviceToHost));
        classIdx = 0;
        for (int j = 1; j < NUM_CLASSES; j++)
        {
            float prob;
            checkCudaErrors(hipMemcpy(&prob, dSoftmaxOutput + j, sizeof(float), hipMemcpyDeviceToHost));
            if (prob > classProb)
            {
                classProb = prob;
                classIdx = j;
            }
        }

        // Update accuracy
        if (classIdx == hTestLabels[i])
        {
            numCorrect++;
        }
    }
    float accuracy = (float)numCorrect / NUM_TEST_IMAGES;
    printf("Accuracy: %f\n", accuracy);

    // Clean up
    checkCudaErrors(hipFree(dTrainingData));
    checkCudaErrors(hipFree(dTestData));
    checkCudaErrors(hipFree(dFcWeights));
    checkCudaErrors(hipFree(dFcBiases));
    checkCudaErrors(hipFree(dFcInput));
    checkCudaErrors(hipFree(dFcOutput));
    checkCudaErrors(hipFree(dFcOutputGrad));
    checkCudaErrors(hipFree(dSoftmaxOutput));
    checkCudaErrors(hipFree(dSoftmaxLoss));
    checkCudaErrors(hipFree(dSoftmaxOutputGrad));
    checkCudaErrors(hipFree(dConvFwdWorkspace));
    checkCudaErrors(hipFree(dConvBwdDataWorkspace));
    checkCudaErrors(hipFree(dConvBwdFilterWorkspace));
    checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensor));
    checkCUDNN(hipdnnDestroyFilterDescriptor(filterDesc));
    checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
    checkCudaErrors(hipblasDestroy(cublas));
    checkCUDNN(hipdnnDestroy(cudnn));

    delete[] hTrainingData;
    delete[] hTestData;
    delete[] hTestLabels;
    delete[] hFcWeights;
    delete[] hFcBiases;
    
    return 0;
}
   



