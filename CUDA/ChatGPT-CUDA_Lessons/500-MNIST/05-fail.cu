#include <hipDNN.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdint.h>

#define IMAGE_WIDTH 28
#define IMAGE_HEIGHT 28
#define IMAGE_SIZE IMAGE_WIDTH * IMAGE_HEIGHT
#define NUM_CLASSES 10
#define NUM_CHANNELS 1
#define BATCH_SIZE 64
#define NUM_EPOCHS 10
#define LEARNING_RATE 0.001f

const int HIDDEN_SIZE = 128;

struct TrainingData
{
    float image[IMAGE_SIZE];
    float label[NUM_CLASSES];
};

struct TestData
{
    float image[IMAGE_SIZE];
    float label[NUM_CLASSES];
};

int32_t readInt(std::ifstream& stream)
{
    int32_t value;
    stream.read((char*)&value, 4);
    return __builtin_bswap32(value);
}

// Load MNIST training data
int loadTrainingData(TrainingData* trainingData)
{
    // Open the training images file
    std::ifstream file("../../../zData/MNIST/train-images-idx3-ubyte", std::ios::binary);

    // Read the magic number and number of images
    int32_t magic = readInt(file);
    int32_t numImages = readInt(file);

    // Read the image data
    for (int i = 0; i < numImages; i++)
    {
        for (int j = 0; j < IMAGE_SIZE; j++)
        {
            uint8_t pixel;
            file.read((char*)&pixel, 1);
            hTrainingData[i * IMAGE_SIZE + j] = pixel / 255.0f;
    }
    
    // Open training labels file
    std::ifstream file("../../../zData/MNIST/train-labels-idx1-ubyte", std::ios::binary);
    if (!file.is_open())
    {
        std::printf("Error opening file\n");
        return 1;
    }

    // Read header
    int32_t magicNumber;
    int32_t numLabels;
    file.read((char*)&magicNumber, 4);
    file.read((char*)&numLabels, 4);
    magicNumber = __builtin_bswap32(magicNumber);
    numLabels = __builtin_bswap32(numLabels);

    // Read data
    for (int i = 0; i < numLabels; i++)
    {
        unsigned char label;
        file.read((char*)&label, 1);
        trainingData[i].label[(int)label] = 1.0f;
    }

    file.close();
    
    // Read the image data
    float* hTrainingData = new float[numImages * IMAGE_SIZE];
    for (int i = 0; i < numImages; i++)
    {
        for (int j = 0; j < IMAGE_SIZE; j++)
        {
            uint8_t pixel;
            file.read((char*)&pixel, 1);
            hTrainingData[i * IMAGE_SIZE + j] = pixel / 255.0f;
        }
    }
    
    // Close the file
    file.close();

    // Open training labels file
    std::ifstream file("../../../zData/MNIST/train-labels-idx1-ubyte", std::ios::binary);
    if (!file.is_open())
    {
        std::printf("Error opening file\n");
        return 1;
    }

    // Read header
    int32_t magicNumber = readInt(file);
    int32_t numLabels = readInt(file);

    // Read data
    for (int i = 0; i < numLabels; i++)
    {
        unsigned char label;
        file.read((char*)&label, 1);
        trainingData[i].label[(int)label] = 1.0f;
    }

    file.close();

    // Copy data to device
    float* dTrainingData;
    hipMalloc((void**)&dTrainingData, numImages * IMAGE_
    
    // Allocate device memory for training data
    checkCudaErrors(hipMalloc((void**)&dTrainingData, NUM_IMAGES * IMAGE_SIZE * sizeof(float)));

    // Copy data to device
    checkCudaErrors(hipMemcpy(dTrainingData, hTrainingData, NUM_IMAGES * IMAGE_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Free host memory
    delete[] hTrainingData;

    // Create device memory for labels
    float* dLabels;
    checkCudaErrors(hipMalloc((void**)&dLabels, NUM_IMAGES * NUM_CLASSES * sizeof(float)));

    // Copy labels to device
    checkCudaErrors(hipMemcpy(dLabels, trainingData, NUM_IMAGES * NUM_CLASSES * sizeof(float), hipMemcpyHostToDevice));

    // Create CUDNN tensors
    hipdnnTensorDescriptor_t inputTensor;
    hipdnnTensorDescriptor_t outputTensor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
    checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input
    
        hipMalloc((void**)&dTrainingData, NUM_IMAGES * IMAGE_SIZE * sizeof(float));
    hipMemcpy(dTrainingData, hTrainingData, NUM_IMAGES * IMAGE_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Copy labels to device
    float* dTrainingLabels;
    hipMalloc((void**)&dTrainingLabels, NUM_IMAGES * NUM_CLASSES * sizeof(float));
    hipMemcpy(dTrainingLabels, trainingData, NUM_IMAGES * NUM_CLASSES * sizeof(float), hipMemcpyHostToDevice);

    // Create input and output tensors
    hipdnnTensorDescriptor_t inputTensorDesc;
    hipdnnTensorDescriptor_t outputTensorDesc;
    hipdnnCreateTensorDescriptor(&inputTensorDesc);
    hipdnnSetTensor4dDescriptor(inputTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, BATCH_SIZE, NUM_CHANNELS, IMAGE_HEIGHT, IMAGE_WIDTH);
    hipdnnCreateTensorDescriptor(&outputTensorDesc);
    hipdnnSetTensor4dDescriptor(outputTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, BATCH_SIZE, NUM_CLASSES, 1, 1);

    // Create filter and bias tensors
    hipdnnFilterDescriptor_t filterDesc;
    hipdnnTensorDescriptor_t biasDesc;
    hipdnnCreateFilterDescriptor(&filterDesc);
    hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, NUM_CLASSES, NUM_CH

// Allocate device memory for labels
float* dLabels;
hipMalloc((void**)&dLabels, NUM_TRAINING_IMAGES * NUM_CLASSES * sizeof(float));

// Copy labels to device
hipMemcpy(dLabels, hLabels, NUM_TRAINING_IMAGES * NUM_CLASSES * sizeof(float), hipMemcpyHostToDevice);

// Define layer sizes
const int inputSize = IMAGE_SIZE;
const int hiddenSize = 100;
const int outputSize = NUM_CLASSES;

// Allocate device memory for weights and biases
float* dWeights1, dBiases1, dWeights2, dBiases2;
hipMalloc((void)&dWeights1, inputSize * hiddenSize * sizeof(float));
hipMalloc((void*)&dBiases1, hiddenSize * sizeof(float));
hipMalloc((void**)&dWeights2, hiddenSize * outputSize * sizeof(float));
hipMalloc((void**)&dBiases2, outputSize * sizeof(float));

// Initialize weights and biases with random values
initializeWeights(dWeights1, inputSize * hiddenSize);
initializeWeights(dBiases1, hiddenSize);
initializeWeights(dWeights2, hiddenSize * outputSize);
initializeWeights(dBiases2, outputSize);

// Allocate device memory for layer inputs and outputs
float* dInput, dHidden, dOutput;
hipMalloc((void)&dInput, BATCH_SIZE * inputSize * sizeof(float));
hipMalloc((void**)&dHidden, BATCH_SIZE * hiddenSize * sizeof(float));
hipMalloc((void**)&dOutput, BATCH_SIZE * outputSize * sizeof(float));

// Allocate memory on device
hipMalloc((void**)&dTrainingData, NUM_IMAGES * IMAGE_SIZE * sizeof(float));
hipMalloc((void**)&dTrainingLabels, NUM_IMAGES * NUM_CLASSES * sizeof(float));

// Copy data to device
hipMemcpy(dTrainingData, hTrainingData, NUM_IMAGES * IMAGE_SIZE * sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(dTrainingLabels, hTrainingLabels, NUM_IMAGES * NUM_CLASSES * sizeof(float), hipMemcpyHostToDevice);

// Free host memory
delete[] hTrainingData;
delete[] hTrainingLabels;

// Set up input tensor descriptor
hipdnnTensorDescriptor_t inputTensorDesc;
hipdnnCreateTensorDescriptor(&inputTensorDesc);
hipdnnSetTensor4dDescriptor(inputTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, BATCH_SIZE, NUM_CHANNELS, IMAGE_WIDTH, IMAGE_HEIGHT);

// Set up output tensor descriptor
hipdnnTensorDescriptor_t outputTensorDesc;
hipdnnCreateTensorDescriptor(&outputTensorDesc);
hipdnnSetTensor4dDescriptor(outputTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, BATCH_SIZE, NUM_CLASSES, 1, 1);

// Set up convolution layer
hipdnnConvolutionDescriptor_t convDesc;
hipdnnCreateConvolutionDescriptor(&convDesc);
hipdnnSetConvolution2dDescriptor(convDesc,
    
// Copy data to device
float* dTrainingData;
hipMalloc((void**)&dTrainingData, NUM_IMAGES * IMAGE_SIZE * sizeof(float));
hipMemcpy(dTrainingData, hTrainingData, NUM_IMAGES * IMAGE_SIZE * sizeof(float), hipMemcpyHostToDevice);

// Create device arrays for labels
float* dLabels;
hipMalloc((void**)&dLabels, NUM_IMAGES * NUM_CLASSES * sizeof(float));
hipMemcpy(dLabels, trainingData, NUM_IMAGES * NUM_CLASSES * sizeof(float), hipMemcpyHostToDevice);

// Create device arrays for weights and biases
float* dWeights1;
float* dBias1;
float* dWeights2;
float* dBias2;
hipMalloc((void**)&dWeights1, IMAGE_SIZE * HIDDEN_SIZE * sizeof(float));
hipMalloc((void**)&dBias1, HIDDEN_SIZE * sizeof(float));
hipMalloc((void**)&dWeights2, HIDDEN_SIZE * NUM_CLASSES * sizeof(float));
hipMalloc((void**)&dBias2, NUM_CLASSES * sizeof(float));

// Initialize weights and biases to random values
srand(time(NULL));
for (int i = 0; i < IMAGE_SIZE * HIDDEN_SIZE; i++)
{
dWeights1[i] = (float)rand() / RAND_MAX;
}
for (int i = 0; i < HIDDEN_SIZE; i++)
{
dBias1[i] = (float)rand() / RAND_MAX;
}
for

// Allocate device memory for labels
float* dTrainingLabels;
hipMalloc((void**)&dTrainingLabels, NUM_TRAINING_IMAGES * NUM_CLASSES * sizeof(float));

// Copy labels to device
hipMemcpy(dTrainingLabels, hTrainingLabels, NUM_TRAINING_IMAGES * NUM_CLASSES * sizeof(float), hipMemcpyHostToDevice);

// Create hipdnnTensorDescriptor_t for data
hipdnnTensorDescriptor_t dataDesc;
hipdnnCreateTensorDescriptor(&dataDesc);
hipdnnSetTensor4dDescriptor(dataDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, BATCH_SIZE, NUM_CHANNELS, IMAGE_HEIGHT, IMAGE_WIDTH);

// Create hipdnnTensorDescriptor_t for labels
hipdnnTensorDescriptor_t labelDesc;
hipdnnCreateTensorDescriptor(&labelDesc);
hipdnnSetTensor4dDescriptor(labelDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, BATCH_SIZE, NUM_CLASSES, 1, 1);

// Create hipdnnFilterDescriptor_t for weights
hipdnnFilterDescriptor_t weightDesc;
hipdnnCreateFilterDescriptor(&weightDesc);
hipdnnSetFilter4dDescriptor(weightDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, NUM_HIDDEN, NUM_CHANNELS, KERNEL_WIDTH, KERNEL_HEIGHT

// NOTE - At this point I give up on this one too. This sort of thing happens a lot. I used to be able to argue it in to finally working. But on Dec 28 2022 they seem to be limiting hourly questions.

#include <hipDNN.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define IMAGE_WIDTH 28
#define IMAGE_HEIGHT 28
#define IMAGE_SIZE IMAGE_WIDTH * IMAGE_HEIGHT
#define NUM_CLASSES 10
#define NUM_CHANNELS 1
#define BATCH_SIZE 64
#define NUM_EPOCHS 10
#define LEARNING_RATE 0.001f

struct TrainingData
{
float image[IMAGE_SIZE];
float label[NUM_CLASSES];
};

struct TestData
{
float image[IMAGE_SIZE];
float label[NUM_CLASSES];
};

int32_t readInt(std::ifstream& stream)
{
int32_t value;
stream.read((char*)&value, 4);
return __builtin_bswap32(value);
}

// Load MNIST training data
int loadTrainingData(TrainingData* trainingData)
{
// Open the training images file
std::ifstream file("../../../zData/MNIST/train-images-idx3-ubyte", std::ios::binary);

// Read the magic number and number of images
int32_t magic = readInt(file);
int32_t numImages = readInt(file);

// Read the image data
float* hTrainingData = new float[numImages * IMAGE_SIZE];
for (int i = 0; i < numImages; i++)
{
    for (

